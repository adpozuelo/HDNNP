#include "hip/hip_runtime.h"
/*
  UOC_TFG - Antonio Díaz Pozuelo - adpozuelo@uoc.edu
  Start neural network functions file
  Start neural network epochs and optimizing (learning process) variables between neurons and bias
  Header is specify in nn_start.h
  High-Dimensional Neural Network Potentials (HDNNP)
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <math.h>
#include "messages.h"
#include "conf.h"

// Extern functions
extern"C" {
// wrapper function to call IMSL FORTRAN optimization library
	void wrapper_(int *np, int *nop, int *niter, int *icalc, float *x, float *y, float *yy, int *npar, float *xg, int *ib, float *xlb, float *xub, float *ftol, int *maxfcn, float *xfin, float *fvec, int *noa, int *ils, int *first);
	// residue function (Neural Network epochs)
	void residue_(int *symmetry_functions_number, float *symmetry_functions, float *energies, float *energies_fit, int *energies_number, float *parameters, int *parameters_number,int *iteration,float *residue, int *mode, int *atoms_number, int *input_layer_size);
}

void nn_start(char *input_filename_g_normalized_learn, char *input_filename_energy_learn, char *input_filename_misc_data, char *output_filename_learning_process, char *input_filename_g_normalized_predict, char *output_filename_predict) {

	char *tmpBuffer = (char*) malloc(BUFFER_SIZE * sizeof(char)); // Allocate memory buffer

	// Open G normalized symmetry functions learn file in read mode
	FILE *input_file_g_learn = fopen(input_filename_g_normalized_learn, "r");
	if (input_file_g_learn == NULL) {
		print_opening_file_error_and_exit();
	}
	// Read input layer size
	int input_layer_size;
	if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char), input_file_g_learn) == NULL) {
		print_no_data_in_file_error_and_exit();
	} else {
		input_layer_size = atoi(++tmpBuffer);
	}

	// Energy average, min and max values for las bias parameter
	float energy_average;
	float min_energy;
	float max_energy;

	// Open miscelanea file
	FILE *input_file_misc = fopen(input_filename_misc_data, "r");
	if (input_file_misc == NULL) {
		print_opening_file_error_and_exit();
	}
	if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char), input_file_misc) == NULL) {
		print_no_data_in_file_error_and_exit();
	} else {
		energy_average = atof(tmpBuffer); // Read energy average
	}
	if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char), input_file_misc) == NULL) {
		print_no_data_in_file_error_and_exit();
	} else {
		int n = 0;
		char *token;
		token = strtok(tmpBuffer, DELIM);
		while (token) {
			if (n == 0) // First token is energy minimum value                                                  
				min_energy = atof(token);
			if (n == 1) // Second token is energy maximum value                                                  
				max_energy = atof(token);
			n++;
			token = strtok('\0', DELIM);
		}
	}
	// Control total (learn and predict) timesteps
	int total_learn_timesteps = 0; 
	if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char), input_file_misc) == NULL) {
		print_no_data_in_file_error_and_exit();
	} else {
		total_learn_timesteps = atoi(tmpBuffer); // Read learn timesteps
	}
	int total_predict_timesteps = 0;
	if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char), input_file_misc) == NULL) {
		print_no_data_in_file_error_and_exit();
	} else {
		total_predict_timesteps = atoi(tmpBuffer); // Read predict timesteps
	}
	// Control number of atoms per box
	int number_of_atoms_per_box = 0;
	if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char), input_file_misc) == NULL) {
		print_no_data_in_file_error_and_exit();
	} else {
		number_of_atoms_per_box = atoi(tmpBuffer); // Read number of atoms per box
	}
	fclose(input_file_misc); // Close miscelanea file

	/*
	  Neural network parameters
	*/
	int parameters_number = ((input_layer_size * HIDDEN_LAYER_SIZE) + HIDDEN_LAYER_SIZE) +
		((HIDDEN_LAYER_SIZE * HIDDEN_LAYER_SIZE) + HIDDEN_LAYER_SIZE) + HIDDEN_LAYER_SIZE + 1;
	float parameters[parameters_number]; // Initial parameters (at last it will be fitted)
	float parameters_fit[parameters_number]; // Fitted parameters
	// Initialize parameters with random number (-1, 1)
	srand(time(NULL));
	float a = 2.0;
	for (int i = 0; i < parameters_number; i++){
		parameters[i] = (float) (rand() / (float) ((RAND_MAX)) * a) - 1;
	}
	// Lowest and highest parameters values
	float xlb[parameters_number];
	float xub[parameters_number];
	// Initialize lowest and highest values (-1, 1)
	for (int i = 0; i < parameters_number - 1; i++){
		xlb[i]=-1;
		xub[i]=1;
	}
	/* 
	   Last parameter is bias weight to output neuron guide it to correct optimization.
	   It is initialized with energy average and its limits are bound to maximum and minimum energies values
	*/
	xlb[parameters_number-1] = min_energy / number_of_atoms_per_box;
	xub[parameters_number-1] = max_energy / number_of_atoms_per_box;
	parameters[parameters_number-1] = energy_average / number_of_atoms_per_box;

	/*
	  IMSL parameters
	*/
	float ftol = FTOL; // OPT tolerance
	int ib = IB; // OPT option
	int maxfcn = MAXFCN; // OPT maximum functions to call
	int mode; // Neural network mode: 1 to learn, 0 to predict
	float fvec = 0; // Cost function's residue for IMSL optimization
	int nop = NUMBER_OF_BOXES_TO_OPT; // Number of boxes to optimize in paralell mode
	int first_exe = 1; // First wrapper execution control to IWKIN FORTRAN parameter
	
	/*
	  Energies
	*/
	float energies[NUMBER_OF_BOXES_TO_OPT]; // Supervised energies
	float energies_fit[NUMBER_OF_BOXES_TO_OPT]; // Predicted energies
	// Open energy learn file in read mode
	FILE *input_file_energy_learn = fopen(input_filename_energy_learn, "r");
	if (input_file_energy_learn == NULL) {
		print_opening_file_error_and_exit();
	}
	
	/*
	  Symmetry functions (input layer)
	*/
	int symmetry_functions_number = input_layer_size * number_of_atoms_per_box * NUMBER_OF_BOXES_TO_OPT;
	float symmetry_functions[symmetry_functions_number];

	// File control variables
	int local_timestep = 0; // Local timesteps counter
	bool file_end = false; // End line control
	
	// Open learning process output file (graph)
	FILE *output_file_learning_process = fopen(output_filename_learning_process, "w");
	if (output_file_learning_process == NULL) {
		print_opening_file_error_and_exit();
	}
	// Write header to learning process output file
	fprintf(output_file_learning_process, "Energy_Predicted Energy_Supervised\n");

	// Print process information
	printf("Neural network start\nNeural network learning!\n");
	
	/*
	  Neural network learning mode
	 */
	while (!file_end && local_timestep < total_learn_timesteps) { // Go over all input learn file
	 	int niter = 0; // Local iterations for ISML optimization
		for (int j = 0; j < NUMBER_OF_BOXES_TO_OPT; j++) { // Go over number of boxes to process and optimize
			for (int i = 0; i < number_of_atoms_per_box; i++) { // Read atoms from G normalized learn file
				if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char),
						  input_file_g_learn) == NULL) {
					file_end = true;
				} else {
					int n = 0;
					char *token;
					token = strtok(tmpBuffer, DELIM);
					while (token && n < input_layer_size) {
						// read each representation value of atom and store them into vector (serialized data)
						symmetry_functions[(j * number_of_atoms_per_box * input_layer_size) +
										   (i * input_layer_size) + n] = atof(token);
						n++;
						token = strtok('\0', DELIM);
					}
				}
			}
			// Read energy from learn file
			if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char),
					  input_file_energy_learn) == NULL) {
				file_end = true;
			} else {
				energies[j] = atof(tmpBuffer); // Read energy
			}
		}
		// Neural network in learning process start
		if (!file_end) {
			// Call to FORTRAN WRAPPER (NN in learning mode)
			wrapper_(&symmetry_functions_number, &nop, &niter, &mode, symmetry_functions, energies, energies_fit, &parameters_number, parameters, &ib, xlb, xub, &ftol, &maxfcn, parameters_fit, &fvec, &number_of_atoms_per_box, &input_layer_size, &first_exe); 
			// Write learning process to output file (graph)
			for (int i = 0; i < nop; i++)
				fprintf(output_file_learning_process, "%.10e %.10e\n", energies_fit[i], energies[i]);
			fflush(output_file_learning_process); // Flush file stream

			local_timestep+=NUMBER_OF_BOXES_TO_OPT; // Update timestep local counter
			first_exe = 0; // Set first execution control variable to 0
		}
		// Reading control file *** REMOVE ***
		// if (local_timestep == 500)
		// 	file_end=true;
		// Reading control file *** REMOVE ***
	}
	// Closing learning files
	fclose(input_file_g_learn);
	fclose(input_file_energy_learn);
	fclose(output_file_learning_process);

	// Open G normalized symmetry functions predict file in read mode
	FILE *input_file_g_predict = fopen(input_filename_g_normalized_predict, "r");
	if (input_file_g_predict == NULL) {
		print_opening_file_error_and_exit();
	}
  
	// Open output predict file in write mode 
	FILE *output_file_predict = fopen(output_filename_predict, "w");
	if (output_file_predict == NULL) {
		print_opening_file_error_and_exit();
	}

	// Print process information
	printf("\nNeural network predicting!\n");
	
	local_timestep = 0; // Reset local timesteps
	file_end = false; // Reset file end control
	/*
	  Neural network predicting mode
	 */
	while (!file_end && local_timestep < total_predict_timesteps) { // Go over all input predict file
		for (int j = 0; j < NUMBER_OF_BOXES_TO_OPT; j++) { // Go over number of boxes to process and optimize
			for (int i = 0; i < number_of_atoms_per_box; i++) { // Read atoms from G normalized learn file
				if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char),
						  input_file_g_predict) == NULL) {
					file_end = true;
				} else {
					int n = 0;
					char *token;
					token = strtok(tmpBuffer, DELIM);
					while (token && n < input_layer_size) {
						// read each representation value of atom and store them into vector (serialized data)
						symmetry_functions[(j * number_of_atoms_per_box * input_layer_size) +
										   (i * input_layer_size) + n] = atof(token);
						n++;
						token = strtok('\0', DELIM);
					}
				}
			}
		}
		// Neural network in predicting process start
		if (!file_end) {
			mode = 0; // Set mode to NN prediction
			int niter = 0; // Reset local iteration
			// Call to GPU neural network epoch in predict mode
			residue_(&symmetry_functions_number, symmetry_functions, energies, energies_fit, &nop, parameters, &parameters_number, &niter, &fvec, &mode, &number_of_atoms_per_box, &input_layer_size);
			// Write energies predicted to output energies file
			for (int i=0; i<nop; i++)
				fprintf(output_file_predict, "%.10e\n", energies_fit[i]);
			fflush(output_file_predict); // Flush file stream

			local_timestep+=NUMBER_OF_BOXES_TO_OPT; // Update timestep local counter
		}
		// Reading control file *** REMOVE ***
		// if (local_timestep == 500)
		// 	file_end=true;
		// Reading control file *** REMOVE ***
	}
  
	// Closing predict files
	fclose(input_file_g_predict);
	fclose(output_file_predict);

	// Print process information
	printf("Neural network stop\n");
}
