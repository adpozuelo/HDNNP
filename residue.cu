#include "hip/hip_runtime.h"
/*
  UOC_TFG - Antonio Díaz Pozuelo - adpozuelo@uoc.edu
  Neural network epochs functions file
  Execute neural network epochs
  Header is specify in residue.h
  High-Dimensional Neural Network Potentials (HDNNP)
*/

#include <stdio.h>
#include <math.h>
#include "conf.h"

// GPU function to calculate sigmoid (neural stimulation)
__device__ float sigmoid(float value) {
	return (float) 1 / (1 + __expf(-value));
}

// GPU function to execute neural network epoch
__global__ void nn_epoch_gpu(float *dev_symmetry_functions, float *dev_energies_fit, float *dev_parameters, const int symmetry_functions_number, const int input_layer_size, const int parameters_number, const int atoms_number) { 

	int tid = blockIdx.x; // Block of atoms (every box is a box of atoms)
	int tjd = threadIdx.x; // Atom inside box of atoms (every thread is a atom)
	/*
	  Each atoms is a neural network itself.
	  Finally, all atoms (NNs) adds their output neurons (atomic energies) to get total energy
	*/

	__shared__ float atomic_energies[MAX_ATOMS_NUMBER]; // Atomic energies of the box
 
	float input_layer[G_TOTAL_SIZE]; // Atomic neural network input layer
	// Deserialize symmetry functions data vector to get atom input layer
	for (int i=0; i < input_layer_size; i++)
		input_layer[i] = dev_symmetry_functions[(tid * atoms_number * input_layer_size) + (tjd * input_layer_size) + i];

	// BIAS parameters start
	int bias_start = parameters_number - 1 - (2 * HIDDEN_LAYER_SIZE);

	// First neural network hidden layer
	float hidden_layer_one[HIDDEN_LAYER_SIZE];
	// Initialize it to zero
	for (int i = 0; i < HIDDEN_LAYER_SIZE; i++)
		hidden_layer_one[i] = 0;
	// Calculus to stimulated neurons values for first hidden layer
	for (int i = 0; i < HIDDEN_LAYER_SIZE; i++){
		for (int j = 0; j < input_layer_size; j++){
			// Summation input layer values plus parameters
			hidden_layer_one[i] += input_layer[j] * dev_parameters[(i * input_layer_size) + j];
		}
		hidden_layer_one[i] += dev_parameters[bias_start + i]; // Add BIAS parameter
		hidden_layer_one[i] = sigmoid(hidden_layer_one[i]); // Simulate neuron
	}

	// Second hidden layer parameters start
	int second_hidden_layer_parameters_start = input_layer_size * HIDDEN_LAYER_SIZE;
	// Second neural network hidden layer
	float hidden_layer_two[HIDDEN_LAYER_SIZE];
	// Initialize it to zero
	for (int i = 0; i < HIDDEN_LAYER_SIZE; i++)
		hidden_layer_two[i] = 0;
	// Calculus to stimulated neurons values for second hidden layer
	for (int i = 0; i < HIDDEN_LAYER_SIZE; i++){
		for (int j = 0; j < HIDDEN_LAYER_SIZE; j++){
			// Summation first hidden layer neurons values plus parameters
			hidden_layer_two[i] += hidden_layer_one[j] * dev_parameters[second_hidden_layer_parameters_start + (i * HIDDEN_LAYER_SIZE) + j];
		}
		hidden_layer_two[i] += dev_parameters[bias_start + HIDDEN_LAYER_SIZE + i]; // Add BIAS parameter
		hidden_layer_two[i] = sigmoid(hidden_layer_two[i]); // Stimulate neuron
	}

	// Output neuron parameters start
	int output_neuron_parameters_start = second_hidden_layer_parameters_start + (HIDDEN_LAYER_SIZE * HIDDEN_LAYER_SIZE);
	// Initilize output neuron to zero
	float output_neuron = 0;
	// Calculus to stimulated neurons values for output neuron
	for (int i = 0; i < HIDDEN_LAYER_SIZE; i++) {
		// Summation second hidden layer neurons values plus parameters
		output_neuron += hidden_layer_two[i] * dev_parameters[output_neuron_parameters_start + i];
	}
	output_neuron += dev_parameters[parameters_number - 1]; // Add BIAS parameter

	// Each neural network (atoms) assign output neuron value (atomic energy) to box atomic energies
	atomic_energies[tjd] = output_neuron;
	__syncthreads(); // Threads syncronization

	// Use a binary reduction to sum all atomic energies
	int i = atoms_number / 2;
	while (i != 0) {
		if (tjd < i)
			atomic_energies[tjd] = atomic_energies[tjd + i] + atomic_energies[tjd];
		__syncthreads();
		i /= 2;
	}

	// Atom 0 (NN 0) assign box energy to energies fitted (predicted)
	if (tjd == 0) {
		dev_energies_fit[tid] = atomic_energies[0];
	}
}

extern "C" void residue_(int *symmetry_functions_number, float *symmetry_functions, float *energies, float *energies_fit, int *energies_number, float *parameters, int *parameters_number,int *iteration,float *residue, int *mode, int *atoms_number, int *input_layer_size){ 

	static float sum; // Temporal residue
	float sump[*energies_number]; // Cuadratic error energies per box

	// Device (GPU) variables: symmetry functions, energies fitted and parameters
	static float *dev_symmetry_functions, *dev_energies_fit, *dev_parameters;

	/*
	  In the first iteration allocating device memory (GPU)
	  Copy from CPU to GPU symmetry functions (data serialized vector)
	*/
	if (*iteration < 2) {
		hipMalloc( (void**)&dev_symmetry_functions, *symmetry_functions_number * sizeof(float) );
		hipMalloc( (void**)&dev_energies_fit, *energies_number * sizeof(float) );
		hipMalloc( (void**)&dev_parameters, *parameters_number *sizeof(float) );
		hipMemcpy( dev_symmetry_functions, symmetry_functions, *symmetry_functions_number * sizeof(float),hipMemcpyHostToDevice );
	}

	// Copy from CPU to GPU parameters
	hipMemcpy( dev_parameters, parameters, *parameters_number * sizeof(float),hipMemcpyHostToDevice );

	// Kernel Call (GPU Kernel)
	nn_epoch_gpu<<<*energies_number, *atoms_number>>>(dev_symmetry_functions, dev_energies_fit, dev_parameters, *symmetry_functions_number, *input_layer_size, *parameters_number, *atoms_number);
	hipMemcpy(energies_fit, dev_energies_fit, *energies_number *sizeof(float),hipMemcpyDeviceToHost );

	/*
	  Neural network learning mode
	*/
	if (*mode == 1) {
		// Cuadratic error calculus for each box of atoms
		for (int i = 0; i < *energies_number; i++){
			sump[i] = (energies_fit[i] - energies[i]) * (energies_fit[i] - energies[i]);
		}
		sum = 0;
		// Sum all cuadratic errors
		for (int i = 0; i < *energies_number; i++) {
			sum += sump[i];
		}
		// Residue calculus
		sum /= *energies_number;
	} else {      
		/*
		  Neural network predict mode
		 */
		hipFree(dev_symmetry_functions);
		hipFree(dev_parameters);
	}
	*residue = sum; // Assign residue to input argument
}
