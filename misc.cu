#include "hip/hip_runtime.h"
/*
  UOC_TFG - Antonio Díaz Pozuelo - adpozuelo@uoc.edu
  Miscellaneous functions file
  Utility functions without specific area
  Header is specify in misc.h
  High-Dimensional Neural Network Potentials (HDNNP)
*/

#include <regex.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "misc.h"
#include "messages.h"

void application_input_control(int argc, char *argv[], int *number_learning_temperatures_final, int *number_predict_temperatures_final) {

	int number_learning_temperatures = 0; // Local learning temperatures counter
	int number_predict_temperatures = 0; // Local predict temperatures counter

	/*
	  Regular expression allowed, each argument have to:
	  1.- Start with "l" (learn) or "p" (predict)
	  2.- Continue from 1 to 4 digits from 0 to 9 values
	*/
	char pattern[18] = "^(l|p)[0-9]{1,4}$";

	for (int i = 1; i < argc; i++) { // Go over all arguments
		if (regex_match(argv[i], pattern)) { // If argument match previous regular expression
			if (strstr(argv[i], "l") != NULL) { // If string contains "l" (learn)
				number_learning_temperatures++; // Add one to learning temperatures counter
			}
			if (strstr(argv[i], "p") != NULL) { // If string contains "p" (predict)
				number_predict_temperatures++; // Add one to predict temperatures counter
			}
		} else { // Else, string doesn't contain both "l" or "p"
			print_help_and_exit(); // Print help and exit
		}
	}
	// If no learning or predict temperatures, print error message before help and exit
	if (number_learning_temperatures == 0 || number_predict_temperatures == 0) {
		print_both_learning_predict_sets_needed();
		print_help_and_exit();
	}
	// Only one preddict temperature is allowed
	if (number_predict_temperatures > 1) {
		print_only_one_predict_temperature_allowed();
		print_help_and_exit();
	}
	// Update global (passed by reference) both learning and predict temperatures counters
	*number_learning_temperatures_final = number_learning_temperatures;
	*number_predict_temperatures_final = number_predict_temperatures;
}

void generate_filenames(int argc, char *argv[], char *learning_history_files[], char *learning_energy_files[], char *predict_history_files[], char *predict_energy_files[]) {

	int tmp_l = 0; // Local learning temperatures counter
	int tmp_p = 0; // Local predict temperatures counter
	char string_history_begin[17] = "data/HISTORY.Te."; // History filename begin
	char string_energy_begin[11] = "data/E.Te."; // Energy filename begin
	char string_end[2] = "K"; // Both history and energy filenames end

	for (int i = 1; i < argc; i++) { // Go over all arguments
		/*
		  If argument contains "l" (learn) process learning case
		*/
		if (strstr(argv[i], "l") != NULL) {
			char * tmp_str_history_learn; // Local history filename (empty)
			char * tmp_str_energy_learn; // Local energy filename (empty)
			// Allocate memory to both local history and energy filenames
			if ((tmp_str_history_learn = (char*) malloc(
					 strlen(string_history_begin) + strlen(argv[i] + 1)
					 + strlen(string_end) + 1)) != NULL
				&& (tmp_str_energy_learn = (char*) malloc(
						strlen(string_energy_begin) + strlen(argv[i] + 1)
						+ strlen(string_end) + 1)) != NULL) {

				tmp_str_history_learn[0] = '\0'; // Add final string character to local history filename
				// Create history filename as history_begin + argument_temperature + history_end
				strcat(tmp_str_history_learn, string_history_begin);
				strcat(tmp_str_history_learn, argv[i] + 1);
				strcat(tmp_str_history_learn, string_end);
				// Add history filename to global array of history filenames
				learning_history_files[tmp_l] = tmp_str_history_learn;

				tmp_str_energy_learn[0] = '\0'; // Add final string character to local energy filename
				// Create history filename as energy_begin + argument_temperature + energy_end
				strcat(tmp_str_energy_learn, string_energy_begin);
				strcat(tmp_str_energy_learn, argv[i] + 1);
				strcat(tmp_str_energy_learn, string_end);
				// Add energy filename to global array of energy filenames
				learning_energy_files[tmp_l] = tmp_str_energy_learn;
				// Update local learning counter
				tmp_l++;
			} else { // Else, memory allocation fails
				print_memory_allocation_error(); // Print error and exit
			}

		}
		/*
		  If argument contains "p" (predict) process predict case
		  There are no comments because it is the same previous case with same steps
		*/
		if (strstr(argv[i], "p") != NULL) {
			char * tmp_str_history_predict;
			char * tmp_str_energy_predict;
			if ((tmp_str_history_predict = (char*) malloc(
					 strlen(string_history_begin) + strlen(argv[i] + 1)
					 + strlen(string_end) + 1)) != NULL
				&& (tmp_str_energy_predict = (char*) malloc(
						strlen(string_energy_begin) + strlen(argv[i] + 1)
						+ strlen(string_end) + 1)) != NULL) {
				tmp_str_history_predict[0] = '\0';
				strcat(tmp_str_history_predict, string_history_begin);
				strcat(tmp_str_history_predict, argv[i] + 1);
				strcat(tmp_str_history_predict, string_end);
				predict_history_files[tmp_p] = tmp_str_history_predict;

				tmp_str_energy_predict[0] = '\0';
				strcat(tmp_str_energy_predict, string_energy_begin);
				strcat(tmp_str_energy_predict, argv[i] + 1);
				strcat(tmp_str_energy_predict, string_end);
				predict_energy_files[tmp_p] = tmp_str_energy_predict;
				tmp_p++;
			} else {
				print_memory_allocation_error();
			}
		}
	}
}

int regex_match(const char *string, char *pattern) {
	int status; // Status flag
	regex_t re; // Regular expression type
	if (regcomp(&re, pattern, REG_EXTENDED) != 0) { // Compile regular expression and control return value
		print_regex_error_and_exit(); // If compilation fails print error and exit
	}
	status = regexec(&re, string, (size_t) 0, NULL, 0); // Match regular expression in pattern
	regfree(&re); // Free regular expression type
	if (status != 0) { // Regular expression doesn't match in pattern
		return (0); //  Then return 0
	}
	return (1); // Regular expression does match in pattern, then return 1
}

float normalize_float(float value_g, float min, float max) {
	// Return normalized value
	return (float) ((2 * (value_g - min)) / (max - min)) - 1;
}
