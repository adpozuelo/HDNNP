#include "hip/hip_runtime.h"
/*
  UOC_TFG - Antonio Díaz Pozuelo - adpozuelo@uoc.edu
  Symmetry functions file
  Symmetry functions to generate atom's cutoff representation
  Header is specify in symmetry_functions.h
  High-Dimensional Neural Network Potentials (HDNNP)
*/

#include <math.h>
#include <stdio.h>
#include "structs.h"
#include "messages.h"

// GPU function to calculate distance between two atoms into a periodic box
__device__ float atoms_distance_periodic(float c_x_atom1, float c_y_atom1, float c_z_atom1, float c_x_atom2, float c_y_atom2, float c_z_atom2, float long_x, float long_y, float long_z) {

	// Atoms distances calculus
	float diff_x = c_x_atom1 - c_x_atom2;
	float diff_y = c_y_atom1 - c_y_atom2;
	float diff_z = c_z_atom1 - c_z_atom2;
	// Box periodic conditions
	if (diff_x >= 0.5 * long_x)
		diff_x -= long_x;
	else if (diff_x < -0.5 * long_x)
		diff_x += long_x;
	if (diff_y >= 0.5 * long_y)
		diff_y -= long_y;
	else if (diff_y < -0.5 * long_y)
		diff_y += long_y;
	if (diff_z >= 0.5 * long_z)
		diff_z -= long_z;
	else if (diff_z < -0.5 * long_z)
		diff_z += long_z;
	// Return distance between two atoms
	return (float) (diff_x * diff_x) + (diff_y * diff_y) + (diff_z * diff_z);
}

// FC symmetry function
__device__ float fc_symmetry(float r_ij) {
	float rc = 36;
	float result = 0;
	if (r_ij <= rc)
		result = (float) 0.5 * (__cosf((M_PI * r_ij) / rc) + 1);
	return result;
}

// G2 symmetry function
__device__ float g2_symmetry(float n, float r_s, float r_ij) {
	float result = 0;
	result = (float) __expf(-n * ((r_ij - r_s) * (r_ij - r_s))) * fc_symmetry(r_ij);
	return result;
}

// G3 symmetry function
__device__ float g3_symmetry(float n, int s, float l, float r_ij, float r_ik, float r_jk) {
	float result = 0;
	float sqrt_r_ij = __fsqrt_rd(r_ij);
	float sqrt_r_ik = __fsqrt_rd(r_ik);
	float sqrt_r_jk = __fsqrt_rd(r_jk);
	float cos_ijk = (float) ((sqrt_r_ij * sqrt_r_ij) + (sqrt_r_ik * sqrt_r_ik) - (sqrt_r_jk * sqrt_r_jk)) / (2 * sqrt_r_ij * sqrt_r_ik);
	if (cos_ijk < -1.0)
		cos_ijk = -1.0;
	if (cos_ijk > 1.0)
		cos_ijk = 1.0;
	result = (float) __powf((1 + (l * cos_ijk)), s);
	result *= (float) __expf((-n) * ((sqrt_r_ij * sqrt_r_ij) + (sqrt_r_ik * sqrt_r_ik) + (sqrt_r_jk * sqrt_r_jk)));
	result *= (float) fc_symmetry(r_ij) * fc_symmetry(r_ik) * fc_symmetry(r_jk);
	return result;
}

__global__ void kernel_symmetry_functions(G2Combination *dev_g2_combination, G3Combination *dev_g3_combination, Atom *dev_atoms, int *dev_number_of_atoms, float *dev_long_x, float *dev_long_y, float *dev_long_z) {

	int tid = blockIdx.x; // First atom
	int tjd = threadIdx.x; // Second atom
	__shared__ float accumulator; // Accumulator register
	float rc = 36; // Cutoff radius (6A ^ 2)

	for (int i = 0; i < G2_SIZE; i++) { // Go over G2 SF combinations
		float n = dev_g2_combination[i].n;
		float rs = dev_g2_combination[i].rs;
		if (tjd == 0) // Initialize accumulator
			accumulator = 0;
		__syncthreads(); // Threads syncronization
		if (tid != tjd) { // If atoms are different (atom1 and atom2)
			// Calculate distance between atoms (atom1 and atom2)
			float r_ij = atoms_distance_periodic(dev_atoms[tid].c_x, dev_atoms[tid].c_y, dev_atoms[tid].c_z, dev_atoms[tjd].c_x, dev_atoms[tjd].c_y, dev_atoms[tjd].c_z, *dev_long_x, *dev_long_y, *dev_long_z);
			if (r_ij <= rc) { // If atoms are into cutoff
				// Add g2 symmetry function value to accumulator
				atomicAdd(&accumulator, g2_symmetry(n, rs, r_ij));
			}
			__syncthreads();
		}
		// Store G2 accumulator to G2 SF value in atom1 structure
		dev_atoms[tid].g2_symmetry[i] = accumulator;
	}

	for (int i = 0; i < G3_SIZE; i++) { // Go over G3 SF combinations
		float n = dev_g3_combination[i].n;
		int s = dev_g3_combination[i].s;
		float l = dev_g3_combination[i].l;
		if (tjd == 0) // Initialize accumulator
			accumulator = 0;
		__syncthreads();
		if (tid != tjd) { // If atoms are different (atom1 and atom2)
			// Calculate distance between atoms (atom1 and atom2)
			float r_ij = atoms_distance_periodic(dev_atoms[tid].c_x, dev_atoms[tid].c_y, dev_atoms[tid].c_z, dev_atoms[tjd].c_x, dev_atoms[tjd].c_y, dev_atoms[tjd].c_z, *dev_long_x, *dev_long_y, *dev_long_z);
			if (r_ij <= rc) { // If atoms are into cutoff
				for (int k = 0; k < *dev_number_of_atoms; k++) { // Go over all atoms
					if (k != tjd && k != tid) { // If atom3 is different of atom1 and atom2
						// Calculate distance between atoms (atom1 and atom3)
						float r_ik = atoms_distance_periodic(dev_atoms[tid].c_x, dev_atoms[tid].c_y, dev_atoms[tid].c_z, dev_atoms[k].c_x, dev_atoms[k].c_y, dev_atoms[k].c_z, *dev_long_x, *dev_long_y, *dev_long_z);
						if (r_ik <= rc) { // If atoms are into cutoff
							// Calculate distance between atoms (atom2 and atom3)
							float r_jk = atoms_distance_periodic(dev_atoms[tjd].c_x, dev_atoms[tjd].c_y, dev_atoms[tjd].c_z, dev_atoms[k].c_x, dev_atoms[k].c_y, dev_atoms[k].c_z, *dev_long_x, *dev_long_y, *dev_long_z);
							// Add G3 SF value to accumulator
							atomicAdd(&accumulator, g3_symmetry(n, s, l, r_ij, r_ik, r_jk));
						}
						__syncthreads();
					}
				}
			}
		}
		// Store g3 accumulator to g3 symmetry value in atom1 structure
		dev_atoms[tid].g3_symmetry[i] = __powf(2, 1 - s) * accumulator;
	}
}

void calculate_symmetry_functions(Box *box_of_atoms, G2Combination *g2_combination, G3Combination *g3_combination) {

	// Device (GPU) G2 SF combination variables
	G2Combination *dev_g2_combination; // GPU pointer
	hipMalloc((void**) &dev_g2_combination, sizeof(G2Combination) * G2_SIZE); // GPU memory allocation
	hipMemcpy(dev_g2_combination, g2_combination, sizeof(G2Combination) * G2_SIZE, hipMemcpyHostToDevice); // CPU to GPU memory copy

	// Device (GPU) G3 SF combination variables
	G3Combination *dev_g3_combination;
	hipMalloc((void**) &dev_g3_combination, sizeof(G3Combination) * G3_SIZE);
	hipMemcpy(dev_g3_combination, g3_combination, sizeof(G3Combination) * G3_SIZE, hipMemcpyHostToDevice);

	// Device (GPU) box atoms
	Atom *dev_atoms;
	hipMalloc((void**) &dev_atoms, sizeof(Atom) * box_of_atoms->number_of_atoms);
	hipMemcpy(dev_atoms, box_of_atoms->atoms, sizeof(Atom) * box_of_atoms->number_of_atoms, hipMemcpyHostToDevice);

	// Device (GPU) number of atoms in the box
	int *dev_number_of_atoms;
	hipMalloc((void**) &dev_number_of_atoms, sizeof(int));
	hipMemcpy(dev_number_of_atoms, &box_of_atoms->number_of_atoms, sizeof(int), hipMemcpyHostToDevice);

	// Device (GPU) length of the box
	float *dev_long_x, *dev_long_y, *dev_long_z;
	hipMalloc((void**) &dev_long_x, sizeof(float));
	hipMalloc((void**) &dev_long_y, sizeof(float));
	hipMalloc((void**) &dev_long_z, sizeof(float));
	hipMemcpy(dev_long_x, &box_of_atoms->long_x, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_long_y, &box_of_atoms->long_y, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_long_z, &box_of_atoms->long_z, sizeof(float), hipMemcpyHostToDevice);

	// Kernel call (GPU kernel)
	kernel_symmetry_functions<<<box_of_atoms->number_of_atoms, box_of_atoms->number_of_atoms>>>(dev_g2_combination, dev_g3_combination, dev_atoms, dev_number_of_atoms, dev_long_x, dev_long_y, dev_long_z);

	/*
	 * Copy from GPU to CPU memory all atoms of the box with both G2 and G3 SF calculated
	 */
	hipMemcpy(box_of_atoms->atoms, dev_atoms, sizeof(Atom) * box_of_atoms->number_of_atoms, hipMemcpyDeviceToHost);

	// Release GPU memory
	hipFree(dev_g2_combination);
	hipFree(dev_g3_combination);
	hipFree(dev_atoms);
	hipFree(dev_number_of_atoms);
	hipFree(dev_long_x);
	hipFree(dev_long_y);
	hipFree(dev_long_z);
}
