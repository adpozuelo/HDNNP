/*
  UOC_TFG - Antonio Díaz Pozuelo - adpozuelo@uoc.edu
  Messages functions file
  Message control center to possible (future) location text
  Header is specify in messages.h
  High-Dimensional Neural Network Potentials (HDNNP)
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void print_help_and_exit(void) {
	printf("Usage: HDNNP ARGUMENTS...\n");
	printf("Arguments:\n");
	printf("	l   temperature (Kelvin, from 0 to 9999) systems which neural network will learn\n");
	printf("	p   temperature (Kelvin, from 0 to 9999) systems which neural network will predict (only one predict temperature is allowed)\n");
	printf("Examples:\n");
	printf("	HDNNP l456 l654 p333\n");
	printf("	HDNNP l234 l469 l567 l555 p667\n");
	exit(1);
}

void print_both_learning_predict_sets_needed(void) {
	printf("Error: both learning and predict sets are needed!\n");
}

void print_regex_error_and_exit(void) {
	printf("Error: regular expression problem!\n");
	exit(1);
}

void print_memory_allocation_error(void) {
	printf("Error: memory allocation problem!\n");
}

void print_opening_file_error_and_exit(void) {
	printf("File open error");
	exit(1);
}

void print_number_of_atoms_error_and_exit(void) {
	printf("Error: number of atoms have to be the same in all timesteps!\n");
	exit(1);
}

void print_no_data_in_file_error_and_exit(void) {
	printf("Error: no data to read from file!\n");
	exit(1);
}

void print_only_one_predict_temperature_allowed() {
	printf("Error: only one predict temperature is allowed!\n");
}
