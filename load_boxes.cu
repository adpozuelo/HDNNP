#include "hip/hip_runtime.h"
/*
  UOC_TFG - Antonio Díaz Pozuelo - adpozuelo@uoc.edu
  Load boxes and generate SF values functions file
  Read one timestep of atoms to form a box of atoms and calculate and write to files both g2 and g3 symmetry functions for both learn and predict raw data
  Header is specify in load_boxes.h
  High-Dimensional Neural Network Potentials (HDNNP)
*/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "messages.h"
#include "structs.h"
#include "symmetry_functions.h"
#include "conf.h"

void load_boxes_and_generate_symmetry_functions(char *learning_filename, char *predict_filename, int number_of_atoms_per_box, int total_learn_timesteps, int total_predict_timesteps, char *output_filename_g2_learn, char *output_filename_g3_learn, char *output_filename_misc_data, char *output_filename_energy_learn, char *output_filename_g2_predict, char *output_filename_g3_predict, char *output_filename_energy_predict, int *g2_sf_valid, float *g2_sf_min, float *g2_sf_max, int *g3_sf_valid, float *g3_sf_min, float *g3_sf_max) {

	char *tmpBuffer = (char*) malloc(BUFFER_SIZE * sizeof(char)); // Allocate memory to buffer

	printf("\nGenerating learning symmetry functions sets (boxes):");
	/*
	 * G2 symmetry function's variables and their combinations
	 */
	float n[6] = { 0, 0.04, 0.14, 0.32, 0.71, 1.79 };
	float rs[6] = { 0, 1, 2, 3, 4, 5 };
	G2Combination *g2_combination = (G2Combination*) malloc(
		sizeof(G2Combination) * G2_SIZE);
	for (int i = 0; i < 6; i++) { // "n" array variables
		for (int j = 0; j < 6; j++) { // "rs" array variables
			g2_combination[(i * 6) + j].n = n[i];
			g2_combination[(i * 6) + j].rs = rs[j];
		}
	}

	/*
	 * G3 symmetry function's variables and their combinations
	 * "n" previous variable array is used too!!!!
	 */
	int s[4] = { 1, 2, 4, 16 };
	float l[2] = { -1, 1 };
	G3Combination *g3_combination = (G3Combination*) malloc(
		sizeof(G3Combination) * G3_SIZE);
	for (int i = 0; i < 6; i++) { // "n" array variables
		for (int j = 0; j < 4; j++) { // "s" array variables
			for (int k = 0; k < 2; k++) { // "l" array variables
				g3_combination[(i * 8) + (j * 2) + k].n = n[i];
				g3_combination[(i * 8) + (j * 2) + k].s = s[j];
				g3_combination[(i * 8) + (j * 2) + k].l = l[k];
			}
		}
	}

	float global_energy_min = 10E6; // Global minimum energy value for further optimization
	float global_energy_max = -10E6; // Global maximum energy value for further optimization
	float energy_average = 0; // Energy accumulator for further average

	/*
	  G2 and G3 SF accumulator for further feature selection
	 */
	float g2_sf_accumulator[G2_SIZE];
	for (int i=0; i<G2_SIZE; i++){
		g2_sf_accumulator[i]=0;
	}
	float g3_sf_accumulator[G3_SIZE];  
	for (int i=0; i<G3_SIZE; i++){
		g3_sf_accumulator[i]=0;
	}

	Box box_of_atoms; // Box of atoms
	box_of_atoms.number_of_atoms = number_of_atoms_per_box; // Number of atoms per box
	box_of_atoms.atoms = (Atom*) malloc(number_of_atoms_per_box * sizeof(Atom)); // Allocate memory to box of atoms

	// Open leaning file to load boxes in read mode
	FILE *learning_file = fopen(learning_filename, "r");
	if (learning_file == NULL) {
		print_opening_file_error_and_exit();
	}
	// Open G2 symmetry output learn file in write mode
	FILE *output_file_g2_learn = fopen(output_filename_g2_learn, "w");
	if (output_file_g2_learn == NULL) {
		print_opening_file_error_and_exit();
	}
	// Open G3 symmetry output learn file in write mode
	FILE *output_file_g3_learn = fopen(output_filename_g3_learn, "w");
	if (output_file_g3_learn == NULL) {
		print_opening_file_error_and_exit();
	}
	// Write G2 and G3 file headers to output learn files
	fprintf(output_file_g2_learn, "#");
	for (int i = 0; i < 6; i++) {
		for (int j = 0; j < 6; j++) {
			fprintf(output_file_g2_learn, "n=%.2f,rs=%.2f ", n[i], rs[j]);
		}
	}
	fprintf(output_file_g2_learn, "\n");
	fprintf(output_file_g3_learn, "#");
	for (int i = 0; i < 6; i++) {
		for (int j = 0; j < 4; j++) {
			for (int k = 0; k < 2; k++) {
				fprintf(output_file_g3_learn, "n=%.2f,s=%d,l=%2.f ", n[i], s[j], l[k]);
			}
		}
	}
	fprintf(output_file_g3_learn, "\n"); // Add end of line
  
	// Open energy output learn file in write mode
	FILE *output_file_energy_learn = fopen(output_filename_energy_learn, "w");
	if (output_file_energy_learn == NULL) {
		print_opening_file_error_and_exit();
	}

	bool file_end = false; // End line control
	int local_learn_timesteps = 0; // Local timesteps counter

	// While file is not ended and local timestep is less or equal than global learn timesteps
	while (!file_end && local_learn_timesteps < total_learn_timesteps) {
		// Read line (energy line) and control end line case (EOF)
		if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char), learning_file) == NULL) {
			file_end = true;
		} else {
			float energy_temp = atof(tmpBuffer);
			energy_average += energy_temp; // Energy average accumulator
			// Control energy maximum and minimum value
			if (energy_temp > global_energy_max)
				global_energy_max = energy_temp;
			if (energy_temp < global_energy_min)
				global_energy_min = energy_temp;
			// Write box energy to output learn file
			fprintf(output_file_energy_learn, "%s", tmpBuffer);
			fflush(output_file_energy_learn); // Flush file buffer
			local_learn_timesteps++; // Update timestep local counter
		}
		// Read line (box length) and control end line case (EOF)
		if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char), learning_file) == NULL) {
			file_end = true;
		} else {
			int n = 0; // Token counter
			char *token; // Token value
			token = strtok(tmpBuffer, DELIM); // Chopped line in tokens
			while (token) {
				if (n == 0) // First token is x length
					box_of_atoms.long_x = atof(token);
				if (n == 1) // Second token is y length
					box_of_atoms.long_y = atof(token);
				if (n == 2) // Third token is z length
					box_of_atoms.long_z = atof(token);
				n++; // Update token counter
				token = strtok('\0', DELIM);
			}
		}
		for (int i = 0; i < box_of_atoms.number_of_atoms && !file_end; i++) { // Go over all atoms in the box
			// Read line (atom coordinates) and control end line case (EOF)
			if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char),
					  learning_file) == NULL) {
				file_end = true;
			} else {
				int n = 0;
				char *token;
				token = strtok(tmpBuffer, DELIM); // Chopped line in tokens
				while (token) {
					if (n == 0)	// First token is atom's x coordinate
						box_of_atoms.atoms[i].c_x = atof(token);
					if (n == 1) // Second token is atom's y coordinate
						box_of_atoms.atoms[i].c_y = atof(token);
					if (n == 2)	// Third token is atom's z coordinate
						box_of_atoms.atoms[i].c_z = atof(token);
					n++;
					token = strtok('\0', DELIM);
				}
			}
			// Read line (atom forces) and control end line case (EOF)
			if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char),
					  learning_file) == NULL) {
				file_end = true;
			} else {
				int n = 0;
				char *token;
				token = strtok(tmpBuffer, DELIM); // Chopped line in tokens
				while (token) {
					if (n == 0)	// First token is atom's x force
						box_of_atoms.atoms[i].f_x = atof(token);
					if (n == 1) // Second token is atom's y force
						box_of_atoms.atoms[i].f_y = atof(token);
					if (n == 2) // Third token is atom's z force
						box_of_atoms.atoms[i].f_z = atof(token);
					n++;
					token = strtok('\0', DELIM);
				}
			}
		}

		if(!file_end){
			/*
			  calculate symmetry functions for this box of atoms with both G2's and g3's variables combinations
			  !!!! GPU version !!!! 
			*/
			calculate_symmetry_functions(&box_of_atoms, g2_combination, g3_combination);

			// Write G2 symmetry functions in G2 output learn file
			for (int i = 0; i < box_of_atoms.number_of_atoms; i++) {
				for (int j = 0; j < G2_SIZE; j++) {
					// Maximum and minimum control for further normalization
					if (box_of_atoms.atoms[i].g2_symmetry[j] < g2_sf_min[j])
						g2_sf_min[j] = box_of_atoms.atoms[i].g2_symmetry[j];
					if (box_of_atoms.atoms[i].g2_symmetry[j] > g2_sf_max[j])
						g2_sf_max[j] = box_of_atoms.atoms[i].g2_symmetry[j];
					// G2 SF accumulator for further feature selection
					g2_sf_accumulator[j] += box_of_atoms.atoms[i].g2_symmetry[j] * box_of_atoms.atoms[i].g2_symmetry[j];
					// Write G2 SF to output learn G2 file
					fprintf(output_file_g2_learn, "%.10e ",	box_of_atoms.atoms[i].g2_symmetry[j]);
				}
				fprintf(output_file_g2_learn, "\n");
			}

			// write g3 symmetry functions in g3 output file
			for (int i = 0; i < box_of_atoms.number_of_atoms; i++) {
				for (int j = 0; j < G3_SIZE; j++) {
					// maximum and minimum control for further normalization
					if (box_of_atoms.atoms[i].g3_symmetry[j] < g3_sf_min[j])
						g3_sf_min[j] = box_of_atoms.atoms[i].g3_symmetry[j];
					if (box_of_atoms.atoms[i].g3_symmetry[j] > g3_sf_max[j])
						g3_sf_max[j] = box_of_atoms.atoms[i].g3_symmetry[j];
					// G2 SF accumulator for further feature selection
					g3_sf_accumulator[j] += box_of_atoms.atoms[i].g3_symmetry[j] * box_of_atoms.atoms[i].g3_symmetry[j];
					// Write G3 SF to output learn G3 file
					fprintf(output_file_g3_learn, "%.10e ",	box_of_atoms.atoms[i].g3_symmetry[j]);
				}
				fprintf(output_file_g3_learn, "\n");
			}
		}
		// Reading control file *** REMOVE ***
		//if (local_learn_timesteps == 100)
		// file_end = true;
		// Reading control file *** REMOVE ***
	}

	// BEGIN ->> generate graph file with relations between g symmetry functions and forces
	// COMMENT IF DON'T NEED THIS GRAPH
	/*FILE *g_final_output = fopen("data/G_FORCE_RELATION_GRAPH.dat", "w");
	  if (g_final_output == NULL) {
	  print_opening_file_error_and_exit();
	  }
	  for (int i = 0; i < box_of_atoms.number_of_atoms - 1; i++) {
	  for (int j = i + 1; j < box_of_atoms.number_of_atoms; j++) {
	  float force_distance = sqrtf(
	  (box_of_atoms.atoms[i].f_x - box_of_atoms.atoms[j].f_x)
	  * (box_of_atoms.atoms[i].f_x
	  - box_of_atoms.atoms[j].f_x)
	  + (box_of_atoms.atoms[i].f_y
	  - box_of_atoms.atoms[j].f_y)
	  * (box_of_atoms.atoms[i].f_y
	  - box_of_atoms.atoms[j].f_y)
	  + (box_of_atoms.atoms[i].f_z
	  - box_of_atoms.atoms[j].f_z)
	  * (box_of_atoms.atoms[i].f_z
	  - box_of_atoms.atoms[j].f_z));
	  float g_symmetry_distance = 0;
	  for (int k = 0; k < G2_SIZE; k++) {
	  if (g2_sf_valid[k] == true) {
	  g_symmetry_distance += (box_of_atoms.atoms[i].g2_symmetry[k]
	  - box_of_atoms.atoms[j].g2_symmetry[k])
	  * (box_of_atoms.atoms[i].g2_symmetry[k]
	  - box_of_atoms.atoms[j].g2_symmetry[k]);
	  }
	  }
	  for (int k = 0; k < G3_SIZE; k++) {
	  if (g3_sf_valid[k] == true) {
	  g_symmetry_distance += (box_of_atoms.atoms[i].g3_symmetry[k]
	  - box_of_atoms.atoms[j].g3_symmetry[k])
	  * (box_of_atoms.atoms[i].g3_symmetry[k]
	  - box_of_atoms.atoms[j].g3_symmetry[k]);
	  }
	  }
	  g_symmetry_distance = sqrtf(g_symmetry_distance);
	  fprintf(g_final_output, "%.10e %.10e\n", g_symmetry_distance,
	  force_distance);
	  }
	  }
	  fclose(g_final_output);
	  printf("\n\nGraph file G_FORCES relationship \"%s\" generated\n",
	  "data/G_FORCE_RELATION_GRAPH.dat");*/
	// END ->> generate graph file with relations between g symmetry functions and forces
	// COMMENT IF DON'T NEED THIS GRAPH

	// closing learning files files
	fclose(learning_file);
	fclose(output_file_g2_learn);
	fclose(output_file_g3_learn);
	fclose(output_file_energy_learn);

	// Open predict file to load boxes in read mode
	FILE *predict_file = fopen(predict_filename, "r");
	if (predict_file == NULL) {
		print_opening_file_error_and_exit();
	}
	// Open G2 symmetry output predict file in write mode
	FILE *output_file_g2_predict = fopen(output_filename_g2_predict, "w");
	if (output_file_g2_predict == NULL) {
		print_opening_file_error_and_exit();
	}
	// Open G3 symmetry output predict file in write mode
	FILE *output_file_g3_predict = fopen(output_filename_g3_predict, "w");
	if (output_file_g3_predict == NULL) {
		print_opening_file_error_and_exit();
	}
	// Write G2 and G3 file headers
	fprintf(output_file_g2_predict, "#");
	for (int i = 0; i < 6; i++) {
		for (int j = 0; j < 6; j++) {
			fprintf(output_file_g2_predict, "n=%.2f,rs=%.2f ", n[i], rs[j]);
		}
	}
	fprintf(output_file_g2_predict, "\n");
	fprintf(output_file_g3_predict, "#");
	for (int i = 0; i < 6; i++) {
		for (int j = 0; j < 4; j++) {
			for (int k = 0; k < 2; k++) {
				fprintf(output_file_g3_predict, "n=%.2f,s=%d,l=%2.f ", n[i], s[j], l[k]);
			}
		}
	}
	fprintf(output_file_g3_predict, "\n");
  
	// Open energy output predict file in write mode
	FILE *output_file_energy_predict = fopen(output_filename_energy_predict, "w");
	if (output_file_energy_predict == NULL) {
		print_opening_file_error_and_exit();
	}

	file_end = false; // End line control
	int local_predict_timesteps = 0; // Local predict timesteps counter

	// While file is not ended and local timestep is less or equal than global predict timesteps
	while (!file_end && local_predict_timesteps < total_predict_timesteps) {
		// Read line (energy line) and control end line case (EOF)
		if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char), predict_file) == NULL) {
			file_end = true;
		} else {
			// Write box energy to output predict file
			fprintf(output_file_energy_predict, "%s", tmpBuffer);
			fflush(output_file_energy_predict); // Flush file buffer
			local_predict_timesteps++; // Update timestep local counter
		}
		// Read line (box length) and control end line case (EOF)
		if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char), predict_file) == NULL) {
			file_end = true;
		} else {
			int n = 0; // Token counter
			char *token; // Token value
			token = strtok(tmpBuffer, DELIM); // Chopped line in tokens
			while (token) {
				if (n == 0) // First token is x length
					box_of_atoms.long_x = atof(token);
				if (n == 1) // Second token is y length
					box_of_atoms.long_y = atof(token);
				if (n == 2) // Third token is z length
					box_of_atoms.long_z = atof(token);
				n++; // Update token counter
				token = strtok('\0', DELIM);
			}
		}
		for (int i = 0; i < box_of_atoms.number_of_atoms && !file_end; i++) { // Go over all atoms in the box
			// Read line (atom coordinates) and control end line case (EOF)
			if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char),
					  predict_file) == NULL) {
				file_end = true;
			} else {
				int n = 0;
				char *token;
				token = strtok(tmpBuffer, DELIM); // Chopped line in tokens
				while (token) {
					if (n == 0)	// First token is atom's x coordinate
						box_of_atoms.atoms[i].c_x = atof(token);
					if (n == 1) // Second token is atom's y coordinate
						box_of_atoms.atoms[i].c_y = atof(token);
					if (n == 2)	// Third token is atom's z coordinate
						box_of_atoms.atoms[i].c_z = atof(token);
					n++;
					token = strtok('\0', DELIM);
				}
			}
			// Read line (atom forces) and control end line case (EOF)
			if (fgets(tmpBuffer, BUFFER_SIZE * sizeof(char),
					  predict_file) == NULL) {
				file_end = true;
			} else {
				int n = 0;
				char *token;
				token = strtok(tmpBuffer, DELIM); // Chopped line in tokens
				while (token) {
					if (n == 0)	// First token is atom's x force
						box_of_atoms.atoms[i].f_x = atof(token);
					if (n == 1) // Second token is atom's y force
						box_of_atoms.atoms[i].f_y = atof(token);
					if (n == 2) // Third token is atom's z force
						box_of_atoms.atoms[i].f_z = atof(token);
					n++;
					token = strtok('\0', DELIM);
				}
			}
		}

		if (!file_end) {
			/*
			  calculate symmetry functions for this box of atoms with both G2's and g3's variables combinations
			  !!!! GPU version !!!! 
			*/
			calculate_symmetry_functions(&box_of_atoms, g2_combination, g3_combination);
      
			// write g2 symmetry functions in g2 output predict file
			for (int i = 0; i < box_of_atoms.number_of_atoms; i++) {
				for (int j = 0; j < G2_SIZE; j++) {
					// maximum and minimum control for further normalization
					if (box_of_atoms.atoms[i].g2_symmetry[j] < g2_sf_min[j])
						g2_sf_min[j] = box_of_atoms.atoms[i].g2_symmetry[j];
					if (box_of_atoms.atoms[i].g2_symmetry[j] > g2_sf_max[j])
						g2_sf_max[j] = box_of_atoms.atoms[i].g2_symmetry[j];
					// G2 SF accumulator for further feature selection
					g2_sf_accumulator[j] += box_of_atoms.atoms[i].g2_symmetry[j] * box_of_atoms.atoms[i].g2_symmetry[j];
					// Write G2 SF to output learn G2 file
					fprintf(output_file_g2_predict, "%.10e ",	box_of_atoms.atoms[i].g2_symmetry[j]);
				}
				fprintf(output_file_g2_predict, "\n");
			}

			// write g3 symmetry functions in g3 output file
			for (int i = 0; i < box_of_atoms.number_of_atoms; i++) {
				for (int j = 0; j < G3_SIZE; j++) {
					// maximum and minimum control for further normalization
					if (box_of_atoms.atoms[i].g3_symmetry[j] < g3_sf_min[j])
						g3_sf_min[j] = box_of_atoms.atoms[i].g3_symmetry[j];
					if (box_of_atoms.atoms[i].g3_symmetry[j] > g3_sf_max[j])
						g3_sf_max[j] = box_of_atoms.atoms[i].g3_symmetry[j];
					// G3 SF accumulator for further feature selection
					g3_sf_accumulator[j] += box_of_atoms.atoms[i].g3_symmetry[j] * box_of_atoms.atoms[i].g3_symmetry[j];
					// Write G3 SF to output learn G3 file
					fprintf(output_file_g3_predict, "%.10e ",	box_of_atoms.atoms[i].g3_symmetry[j]);
				}
				fprintf(output_file_g3_predict, "\n");
			}
		}
		// Reading control file *** REMOVE ***
		//if (local_predict_timesteps == 100)
		//file_end = true;
		// Reading control file *** REMOVE ***
	}

	// closing predict files
	fclose(predict_file);
	fclose(output_file_g2_predict);
	fclose(output_file_g3_predict);
	fclose(output_file_energy_predict);

	/*
	 * Reducing data dimension (feature selection)
	 */
	float treshold = 1.0E-8; // Vector's modulus threshold

	for (int i = 0; i < G2_SIZE; i++) {
		if (i < 5) { // G2 symmetry functions with n=0 have the same value, remove first forth
			g2_sf_valid[i] = 0;
		} else {    
			g2_sf_accumulator[i] = sqrtf(g2_sf_accumulator[i]); // Modulus acumulator vector
			if (g2_sf_accumulator[i] < treshold) { // If modulus is less than treshold
				g2_sf_valid[i] = 0; // This G2 SF is not valid
			}
		}
	}
	// The same feature selection for G3 SF
	for (int i = 0; i < G3_SIZE; i++) {
		g3_sf_accumulator[i] = sqrtf(g3_sf_accumulator[i]);
		if (g3_sf_accumulator[i] < treshold) {
			g3_sf_valid[i] = 0;
		}
	}

	// Releasing memory
	free(box_of_atoms.atoms);
	free(g2_combination);
	free(g3_combination);
	free(tmpBuffer);

	// Open miscelanea data file (in write mode) to write global values for further normalization and averages
	FILE *misc_data_file = fopen(output_filename_misc_data, "w");
	if (misc_data_file == NULL) {
		print_opening_file_error_and_exit();
	}

	// Write energy average value to miscelanea output file
	fprintf(misc_data_file, "%.10e\n", energy_average / local_learn_timesteps);
	// Write energy maximum and minimum values to miscelanea output file
	fprintf(misc_data_file, "%.10e %.10e\n", global_energy_min, global_energy_max);
	// Write number of atoms per box and both total learn and predict timesteps (boxes) to miscelanea output file
	fprintf(misc_data_file, "%d\n", total_learn_timesteps);
	fprintf(misc_data_file, "%d\n", total_predict_timesteps);
	fprintf(misc_data_file, "%d\n", number_of_atoms_per_box);
	// Close miscelanea output file
	fclose(misc_data_file);

	//print information about timesteps (boxes) processed
	printf("\nLearn symmetry functions sets (boxes) generated: %d\n", local_learn_timesteps);
	printf("Predict symmetry functions sets (boxes) generated: %d\n", local_predict_timesteps);
}
